#include "hip/hip_runtime.h"
#include "ipcv_cuda_adaptive_thresh.cuh"

// CUDA imports
#include <hip/hip_runtime.h>


__global__ void kernelBuildTab(unsigned char* tab, uchar imaxval, int idelta, int type)
{
	int idX = blockDim.x * blockIdx.x + threadIdx.x;
    if (idX < 768)
    {
        if( type == CV_THRESH_BINARY )
            tab[idX] = (uchar)(idX - 255 > -idelta ? imaxval : 0);
        else if( type == CV_THRESH_BINARY_INV )
            tab[idX] = (uchar)(idX - 255 <= -idelta ? imaxval : 0);
        //else // TODO: Throw error
            //CV_Error( CV_StsBadFlag, "Unknown/unsupported threshold type" );
    }
}

    //kernelThreshold<<<dimGrid, dimBlock>>>(deviceSrc, deviceMean, deviceDst, deviceTab, size);

__global__ void kernelThreshold(
    const cv::cuda::PtrStepSz<unsigned char> src,
    const cv::cuda::PtrStepSz<unsigned char> mean,
    cv::cuda::PtrStepSz<unsigned char> dst,
    unsigned char* tab,
    Size size
)
{
	int idX = blockDim.x * blockIdx.x + threadIdx.x;
	int idY = blockDim.y * blockIdx.y + threadIdx.y;

    if (idX < size.width && idY < size.height)
    {
        const uchar* sdata = src.ptr(idY);
        const uchar* mdata = mean.ptr(idY);
        uchar* ddata = dst.ptr(idY);

        ddata[idX] = tab[sdata[idX] - mdata[idX] + 255];
    }
}


// Called from debug.cpp
cv::Mat adaptiveCuda(cv::Mat img, std::string output_path, cv::cuda::Stream _stream) {
    // Binarize the image using OpenCV
    cv::Mat hostBinarized;
    cudaAdaptiveThreshold(img, hostBinarized, 255, ADAPTIVE_THRESH_GAUSSIAN_C, cv::THRESH_BINARY, 5, 2, _stream);

    if (!output_path.empty()) {
        imwrite(output_path, hostBinarized);
#ifdef DIAG
        std::cout << "Image has been written to " << output_path << "\n";
#endif
    }

    return hostBinarized;
}

void cudaAdaptiveThreshold(
    InputArray _src, OutputArray _dst, double maxValue,
    int method, int type, int blockSize, double delta,
    cv::cuda::Stream _stream
){
    Mat host_src = _src.getMat();
    CV_Assert( host_src.type() == CV_8UC1 );
    CV_Assert( blockSize % 2 == 1 && blockSize > 1 );
    Size size = host_src.size();

    // Move src to the GPU.
    cv::cuda::GpuMat dev_src;
    dev_src.upload(host_src);

    _dst.create( size, host_src.type() );
    Mat host_dst = _dst.getMat();
    cv::cuda::GpuMat dev_dst;
    dev_dst.upload(host_dst);

    if( maxValue < 0 )
    {
        host_dst = Scalar(0);
        return;
    }

    Mat host_mean;

    if( host_src.data != host_dst.data )
        host_mean = host_dst;

    if (method == ADAPTIVE_THRESH_MEAN_C)
        // Not implemented. Dunno if it ever will be, bozo.
        CV_Error( CV_StsBadFlag, "Unknown/unsupported adaptive threshold method" );
        /*boxFilter( src, mean, src.type(), Size(blockSize, blockSize),
                   Point(-1,-1), true, BORDER_REPLICATE|BORDER_ISOLATED );*/
    else if (method == ADAPTIVE_THRESH_GAUSSIAN_C)
    {
        /*
        Mat host_srcfloat, host_meanfloat;
        host_src.convertTo(host_srcfloat,CV_32F);

        cv::cuda::GpuMat dev_srcfloat, dev_meanfloat;
        dev_srcfloat.upload(host_srcfloat);
        dev_meanfloat=dev_srcfloat;

        // Gaussian filtering
        cv::Ptr<cv::cuda::Filter> gauss_filter =
            cv::cuda::createGaussianFilter(
                dev_srcfloat.type(),
                -1,
                Size(blockSize, blockSize),
                0,
                0,
                BORDER_REPLICATE,
                -1
            );
        gauss_filter->apply(dev_srcfloat, dev_meanfloat, _stream);

        dev_meanfloat.download(host_meanfloat);
        host_meanfloat.convertTo(host_mean, host_src.type());*/

        // Gaussian filtering
        cv::cuda::GpuMat dev_src, dev_mean;
        dev_src.upload(host_src);
        dev_mean.upload(host_mean);
        cv::Ptr<cv::cuda::Filter> gauss_filter =
            cv::cuda::createGaussianFilter(dev_src.type(), -1, Size(5, 5), 0, 0);
        gauss_filter->apply(dev_src, dev_mean, _stream);
        dev_mean.download(host_mean);
    }
    else
        CV_Error( CV_StsBadFlag, "Unknown/unsupported adaptive threshold method" );

    uchar imaxval = saturate_cast<uchar>(maxValue);
    int idelta = type == THRESH_BINARY ? cvCeil(delta) : cvFloor(delta);
    int magicNumber = 768; // I have literally no idea why this is 768.
    uchar hostTab[768];

    // Allocate space for tab
	unsigned char* deviceTab;
	hipMalloc((void **)&deviceTab, sizeof(unsigned char) * magicNumber);
	//hipMemcpy(deviceTab, hostTab, sizeof(unsigned char) * magicNumber, hipMemcpyHostToDevice);

    // Run kernel to build the tab
    hipStream_t stream =
        cv::cuda::StreamAccessor::getStream(_stream);
	kernelBuildTab<<<1, magicNumber, 0, stream>>>(deviceTab, idelta, imaxval, type);
    hipDeviceSynchronize();

    // Copy finished tab to host (not necessary)
    hipMemcpy(hostTab, deviceTab, sizeof(unsigned char) * magicNumber, hipMemcpyDeviceToHost);

    for (int i = 0; i < magicNumber; i++)
    {
        printf("%d ", hostTab[i]);
    }

    if( host_src.isContinuous() && host_mean.isContinuous() && host_dst.isContinuous() )
    {
        size.width *= size.height;
        size.height = 1;
    }

    // Copy the matricies to the device
    cv::cuda::GpuMat deviceSrc, deviceMean, deviceDst;
    deviceSrc.upload(host_src);
    deviceMean.upload(host_mean);
    deviceDst.upload(host_dst);

    imwrite("/tmp/chom.png", host_src);

    // Set up and run the kernel.
    const int TILE_SIZE = 32;
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((int)ceil((float)size.width / (float)TILE_SIZE), (int)ceil((float)size.height / (float)TILE_SIZE));
    kernelThreshold<<<dimGrid, dimBlock, 0, stream>>>(deviceSrc, deviceMean, deviceDst, deviceTab, size);
    hipDeviceSynchronize();

    // Copy finished product back to host
    deviceDst.download(host_dst);

    
    // Free tab
    hipFree(deviceTab);
}
