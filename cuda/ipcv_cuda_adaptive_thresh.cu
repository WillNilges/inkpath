#include "ipcv_cuda_adaptive_thresh.cuh"

// CUDA imports
#include <hip/hip_runtime.h>

void cudaAdaptiveThreshold( InputArray _src, OutputArray _dst, double maxValue,
                            int method, int type, int blockSize, double delta,
                            cv::cuda::Stream _stream )
{
    Mat src = _src.getMat();
    CV_Assert( src.type() == CV_8UC1 );
    CV_Assert( blockSize % 2 == 1 && blockSize > 1 );
    Size size = src.size();

    _dst.create( size, src.type() );
    Mat dst = _dst.getMat();

    if( maxValue < 0 )
    {
        dst = Scalar(0);
        return;
    }

    Mat mean;

    if( src.data != dst.data )
        mean = dst;

    if (method == ADAPTIVE_THRESH_MEAN_C)
        // Not implemented. Dunno if it ever will be, bozo.
        CV_Error( CV_StsBadFlag, "Unknown/unsupported adaptive threshold method" );
        /*boxFilter( src, mean, src.type(), Size(blockSize, blockSize),
                   Point(-1,-1), true, BORDER_REPLICATE|BORDER_ISOLATED );*/
    else if (method == ADAPTIVE_THRESH_GAUSSIAN_C)
    {
        cv::cuda::GpuMat srcfloat,meanfloat;
        src.convertTo(srcfloat,CV_32F);
        meanfloat=srcfloat;
        //GaussianBlur(srcfloat, meanfloat, Size(blockSize, blockSize), 0, 0, BORDER_REPLICATE|BORDER_ISOLATED);

        // Gaussian filtering
        cv::cuda::GpuMat gpu_blur_in, gpu_blur;
        cv::Ptr<cv::cuda::Filter> gauss_filter =
            cv::cuda::createGaussianFilter(
                srcfloat.type(),
                -1,
                Size(blockSize, blockSize),
                0,
                BORDER_REPLICATE|BORDER_ISOLATED
            );
        gauss_filter->apply(srcfloat, meanfloat, _stream);

        meanfloat.convertTo(mean, src.type());
    }
    else
        CV_Error( CV_StsBadFlag, "Unknown/unsupported adaptive threshold method" );

    int i, j;
    uchar imaxval = saturate_cast<uchar>(maxValue);
    int idelta = type == THRESH_BINARY ? cvCeil(delta) : cvFloor(delta);
    uchar tab[768];

    if( type == CV_THRESH_BINARY )
        for( i = 0; i < 768; i++ )
            tab[i] = (uchar)(i - 255 > -idelta ? imaxval : 0);
    else if( type == CV_THRESH_BINARY_INV )
        for( i = 0; i < 768; i++ )
            tab[i] = (uchar)(i - 255 <= -idelta ? imaxval : 0);
    else
        CV_Error( CV_StsBadFlag, "Unknown/unsupported threshold type" );

    if( src.isContinuous() && mean.isContinuous() && dst.isContinuous() )
    {
        size.width *= size.height;
        size.height = 1;
    }

    for( i = 0; i < size.height; i++ )
    {
        const uchar* sdata = src.ptr(i);
        const uchar* mdata = mean.ptr(i);
        uchar* ddata = dst.ptr(i);

        for( j = 0; j < size.width; j++ )
            ddata[j] = tab[sdata[j] - mdata[j] + 255];
    }
}
