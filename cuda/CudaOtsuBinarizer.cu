#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2020 Dawid Paluchowski

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
 */

#include "CudaOtsuBinarizer.cuh"

#include <stdio.h>

// CUDA imports
#include <hip/hip_runtime.h>

__global__ void kernelCalculateHistogram(unsigned int* histogram, unsigned char* rawPixels, long chunkSize, long totalPixels)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int startPosition = id * chunkSize;
	for (int i = startPosition; i < (startPosition + chunkSize); i++) {
		if (i < totalPixels) {
			int pixelValue = (int)rawPixels[i];
			atomicAdd(&histogram[pixelValue], 1);
		}
	}
}

__global__ void kernelComputeClassVariances(double* histogram, double allProbabilitySum, long int totalPixels, double* betweenClassVariance)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	double firstClassProbability = 0, secondClassProbability = 0;
	double firstClassMean = 0, secondClassMean = 0;
	double firstProbabilitySum = 0;

	for (int t = 0; t <= id % PngImage::MAX_PIXEL_VALUE; t++) {
		firstClassProbability += histogram[t];
		firstProbabilitySum += t * histogram[t];
	}

	secondClassProbability = 1 - firstClassProbability;

	firstClassMean = (double)firstProbabilitySum / (double)firstClassProbability;
	secondClassMean = (double)(allProbabilitySum - firstProbabilitySum) / (double)secondClassProbability;

	betweenClassVariance[id] = firstClassProbability * secondClassProbability * pow((firstClassMean - secondClassMean), 2);
}

__global__ void kernelBinarize(unsigned char* rawPixels, long totalPixels, long chunkSize, unsigned char threshold)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int startPosition = id * chunkSize;
	for (int i = startPosition; i < (startPosition + chunkSize); i++) {
		if (i < totalPixels) {
			if ((int)rawPixels[i] >(int)threshold) {
				rawPixels[i] = PngImage::COLOR_WHITE;
			}
			else {
				rawPixels[i] = PngImage::COLOR_BLACK;
			}
		}
	}
}

CudaOtsuBinarizer::CudaOtsuBinarizer(int threadsPerBlock, int numBlocks, bool drawHistogram, const char* TAG) {
	this->threadsPerBlock_ = threadsPerBlock;
	this->numBlocks_ = numBlocks;
	this->binarizerTimestamp_ = new ExecutionTimestamp();

	this->drawHistogram_ = drawHistogram;
	this->TAG = TAG;
}

CudaOtsuBinarizer::~CudaOtsuBinarizer() {
	delete this->binarizerTimestamp_;
}

PngImage* CudaOtsuBinarizer::binarize(PngImage * imageToBinarize)
{
	long totalImagePixels = (long)imageToBinarize->getRawPixelData().size();

	double* histogram = cudaCalculateHistogram(imageToBinarize->getRawPixelData().data(), totalImagePixels);
	hipDeviceSynchronize();
	
	if (this->drawHistogram_) {
		showHistogram(histogram);
	}

	unsigned char threshold;
	threshold = cudaFindThreshold(histogram, totalImagePixels);
	hipDeviceSynchronize();
	printf("\t[%s] Threshold value: %d\n", this->TAG, threshold);

	delete histogram;

	unsigned char* binarizedRawPixels = cudaBinarize(imageToBinarize->getRawPixelData().data(), totalImagePixels, threshold);
	hipDeviceSynchronize();

	std::vector<unsigned char> binarizedVector(&binarizedRawPixels[0], &binarizedRawPixels[totalImagePixels]);

	delete binarizedRawPixels;

	printf("\n\t[%s] Total calculation time: %.6f milliseconds \n", this->TAG, binarizerTimestamp_->getExecutionTime());

	return new PngImage(
		imageToBinarize->getFilename(),
		imageToBinarize->getWidth(),
		imageToBinarize->getHeight(),
		binarizedVector
	);
}

std::string CudaOtsuBinarizer::getBinarizerExecutionInfo(std::string fileName)
{
	return binarizerTimestamp_->toCommaSeparatedRow(fileName, std::string(this->TAG));
}

void CudaOtsuBinarizer::showHistogram(double* histogram) {
	printf("\nHistogram:\n");
	double value = 0;
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		value = histogram[i];
		printf("\tPixel value %d -> %.5f\n", i, value);
	}
}

double* CudaOtsuBinarizer::cudaCalculateHistogram(unsigned char* rawPixels, long totalPixels) {
	//TODO: check hipGetDeviceProperties function!

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned int* hostHistogram = new unsigned int[PngImage::MAX_PIXEL_VALUE];
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		hostHistogram[i] = 0;
	}

	unsigned int* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceHistogram, hostHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	unsigned char* deviceRawPixels;
	hipMalloc((void **)&deviceRawPixels, sizeof(unsigned char) * totalPixels);
	hipMemcpy(deviceRawPixels, rawPixels, sizeof(unsigned char) * totalPixels, hipMemcpyHostToDevice);

	long chunkSize = ceil(totalPixels / (threadsPerBlock_ * numBlocks_)) + 1;

	hipEventRecord(start);
	kernelCalculateHistogram<<<numBlocks_, threadsPerBlock_>>>(deviceHistogram, deviceRawPixels, chunkSize, totalPixels);
	hipEventRecord(stop);

	hipMemcpy(hostHistogram, deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\t[%s] Histogram calculated in %.6f milliseconds \n", this->TAG, milliseconds);
	binarizerTimestamp_->histogramBuildingTime += milliseconds;

	hipFree(deviceHistogram);
	hipFree(deviceRawPixels);

	double* normalizedHistogram = new double[PngImage::MAX_PIXEL_VALUE];
	long pixelsSum = 0;
	for (int v = 0; v < PngImage::MAX_PIXEL_VALUE; v++) {
		normalizedHistogram[v] = (double)hostHistogram[v] / (double)totalPixels;
		pixelsSum += hostHistogram[v];
	}
	printf("\n\t[%s] Histogram pixels: %d \n", this->TAG, pixelsSum);

	delete hostHistogram;

	return normalizedHistogram;
}

unsigned char CudaOtsuBinarizer::cudaFindThreshold(double* histogram, long int totalPixels) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int threadsPerBlock = 256;
	int numBlocks = 1;

	double allProbabilitySum = 0;
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		allProbabilitySum += i * histogram[i];
	}

	double* hostBetweenClassVariances = new double[PngImage::MAX_PIXEL_VALUE];
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		hostBetweenClassVariances[i] = 0;
	}

	double* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(double) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceHistogram, histogram, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	double* deviceBetweenClassVariances;
	hipMalloc((void **)&deviceBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceBetweenClassVariances, hostBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	hipEventRecord(start);
	kernelComputeClassVariances<<<numBlocks, threadsPerBlock>>>(deviceHistogram, allProbabilitySum, totalPixels, deviceBetweenClassVariances);
	hipEventRecord(stop);
	hipMemcpy(hostBetweenClassVariances, deviceBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\t[%s] Threshold calculated in %.6f milliseconds \n", this->TAG, milliseconds);
	binarizerTimestamp_->thresholdFindingTime += milliseconds;

	hipFree(deviceHistogram);
	hipFree(deviceBetweenClassVariances);

	double maxVariance = 0;
	unsigned char currentBestThreshold = 0;
	for (int t = 0; t < PngImage::MAX_PIXEL_VALUE; t++) {
		if (hostBetweenClassVariances[t] > maxVariance) {
			currentBestThreshold = (unsigned char)t;
			maxVariance = hostBetweenClassVariances[t];
		}
	}

	delete hostBetweenClassVariances;

	return currentBestThreshold;
}

unsigned char* CudaOtsuBinarizer::cudaBinarize(unsigned char * rawPixels, long totalPixels, unsigned char threshold) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned char* hostRawPixels = new unsigned char[totalPixels];

	unsigned char* deviceRawPixels;
	hipMalloc((void **)&deviceRawPixels, sizeof(unsigned char) * totalPixels);
	hipMemcpy(deviceRawPixels, rawPixels, totalPixels * sizeof(unsigned char), hipMemcpyHostToDevice);

	long chunkSize = ceil(totalPixels / (threadsPerBlock_ * numBlocks_)) + 1;

	hipEventRecord(start);
	kernelBinarize<<<numBlocks_, threadsPerBlock_>>>(deviceRawPixels, totalPixels, chunkSize, threshold);
	hipEventRecord(stop);

	hipMemcpy(hostRawPixels, deviceRawPixels, sizeof(unsigned char) * totalPixels, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\t[%s] Binarized in %.6f milliseconds \n", this->TAG, milliseconds);
	binarizerTimestamp_->binarizationTime += milliseconds;

	hipFree(deviceRawPixels);

	return hostRawPixels;
}
