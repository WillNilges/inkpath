#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2020 Dawid Paluchowski

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

===============================================================================

Above is a copy of the license included in https://github.com/palucdev/CudaOtsu/
Portions of this code were copypasta'd from that repo, but all adaptations
made to accomodate OpenCV are my own. Those modifications fall under the
GPL2 license included in ../LICENSE

Cheers,
- Willard

 */

#include "ipcv_cuda.cuh"

#include <stdio.h>

// CUDA imports
#include <hip/hip_runtime.h>

#define M_COORD(i,j) (i*img.cols) + j 
#define MAXPIXVAL 256

// Go through every pixel in the image, and increment its pixel value in the histogram.
__global__ void kernelCalculateHistogram(const cv::cuda::PtrStepSz<unsigned char> input, unsigned int* deviceHistogram)
{
	int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;

    if (idX < input.cols && idY < input.rows) {
        int pixelValue = int(input(idY, idX));
        atomicAdd(&deviceHistogram[pixelValue], 1);
    }
}

// Disgustingly parallel
__global__ void kernelComputeClassVariances(double* histogram, double allProbabilitySum, long int totalPixels, double* betweenClassVariance)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	double firstClassProbability = 0, secondClassProbability = 0;
	double firstClassMean = 0, secondClassMean = 0;
	double firstProbabilitySum = 0;

	for (int t = 0; t <= id % MAXPIXVAL; t++) {
		firstClassProbability += histogram[t];
		firstProbabilitySum += t * histogram[t];
	}

	secondClassProbability = 1 - firstClassProbability;

	firstClassMean = (double)firstProbabilitySum / (double)firstClassProbability;
	secondClassMean = (double)(allProbabilitySum - firstProbabilitySum) / (double)secondClassProbability;

	betweenClassVariance[id] = firstClassProbability * secondClassProbability * pow((firstClassMean - secondClassMean), 2);
}

// Called from debug.cpp
cv::Mat otsuCuda(cv::Mat img, std::string output_path, cv::cuda::Stream _stream) {

    // Gaussian filtering
    // TODO: Play around with this some more (Didn't seem to make much of a difference tbh)
    /*
    cv::cuda::GpuMat gpu_blur;
    cv::Ptr<cv::cuda::Filter> gauss_filter = cv::cuda::createGaussianFilter(img.type(), -1, Size(5, 5), 0, 0);
    gauss_filter->apply(gpu_upsampled, gpu_blur, stream1);
     */

    // Count all the pixels in the image
	long totalImagePixels = (long)img.total();

    // Build an intensity histogram
	double* histogram = cudaCalculateHistogram(img, totalImagePixels, _stream);
	hipDeviceSynchronize();

    // Use the histogram to find the optimal threshold (lowest inter-class variance)
	unsigned char threshold;
	threshold = cudaFindThreshold(histogram, totalImagePixels, _stream);
	hipDeviceSynchronize();

    // Binarize the image using OpenCV
    cv::Mat hostBinarized;
    cv::cuda::GpuMat deviceBinarized;
    deviceBinarized.upload(img);
    cv::cuda::threshold(deviceBinarized, deviceBinarized, (double) threshold, MAXPIXVAL-1, cv::THRESH_BINARY, _stream);
    deviceBinarized.download(hostBinarized);

    if (!output_path.empty()) {
        imwrite(output_path, hostBinarized);
#ifdef DIAG
        std::cout << "Image has been written to " << output_path << "\n";
#endif
    }

    return hostBinarized;
}

double* cudaCalculateHistogram(
        cv::InputArray _input,
        long totalPixels,
        cv::cuda::Stream _stream
){
    // Create a blank array, representing a histogram
    unsigned int* hostHistogram = new unsigned int[MAXPIXVAL];
    for (int i = 0; i < MAXPIXVAL; i++) {
		hostHistogram[i] = 0;
	}

    // Copy histogram to device
	unsigned int* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(unsigned int) * MAXPIXVAL);
	hipMemcpy(deviceHistogram, hostHistogram, sizeof(unsigned int) * MAXPIXVAL, hipMemcpyHostToDevice);

    // Copy image to device
    cv::cuda::GpuMat input;
    input.upload(_input);

    const int TILE_SIZE = 32;
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((int)ceil((float)input.size().width / (float)TILE_SIZE), (int)ceil((float)input.size().height / (float)TILE_SIZE));

    hipStream_t stream =
        cv::cuda::StreamAccessor::getStream(_stream);
    kernelCalculateHistogram<<<dimGrid, dimBlock, 0, stream>>>(input, deviceHistogram);

    // Check for device error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Device error: %s\n", hipGetErrorString(err));
        hipFree(deviceHistogram);
        return nullptr;
    }
    
    // Copy the histogram back to the host
	hipMemcpy(hostHistogram, deviceHistogram, sizeof(unsigned int) * MAXPIXVAL, hipMemcpyDeviceToHost);

    // Free the device Histogram
    hipFree(deviceHistogram);

    // Normalize the Histogram
	double* normalizedHistogram = new double[MAXPIXVAL];
	long pixelsSum = 0;
	for (int v = 0; v < MAXPIXVAL; v++) {
		normalizedHistogram[v] = (double)hostHistogram[v] / (double)totalPixels;
		pixelsSum += hostHistogram[v];
	}
    return normalizedHistogram;
}

// Embarrassingly parallel shit right here
// Pretty much just copypasta'd
unsigned char cudaFindThreshold(double* histogram, long int totalPixels, cv::cuda::Stream _stream)
{
    // Set up kernel (this is a quick one)
	int threadsPerBlock = 256;
	int numBlocks = 1;

    // Total up all the probablilities?
	double allProbabilitySum = 0;
	for (int i = 0; i < MAXPIXVAL; i++) {
		allProbabilitySum += i * histogram[i];
	}

    // Set up array to hold the variances on host
	double* hostBetweenClassVariances = new double[MAXPIXVAL];
	for (int i = 0; i < MAXPIXVAL; i++) {
		hostBetweenClassVariances[i] = 0;
	}

    // Copy histogram to device again
    // TODO: Optimize this? Does it even matter? It's hardly any data. 
	double* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(double) * MAXPIXVAL);
	hipMemcpy(deviceHistogram, histogram, sizeof(double) * MAXPIXVAL, hipMemcpyHostToDevice);

    // Copy variance array to device
	double* deviceBetweenClassVariances;
	hipMalloc((void **)&deviceBetweenClassVariances, sizeof(double) * MAXPIXVAL);
	hipMemcpy(deviceBetweenClassVariances, hostBetweenClassVariances, sizeof(double) * MAXPIXVAL, hipMemcpyHostToDevice);

    // Perform computation
    hipStream_t stream =
        cv::cuda::StreamAccessor::getStream(_stream);
	kernelComputeClassVariances<<<numBlocks, threadsPerBlock>>>(deviceHistogram, allProbabilitySum, totalPixels, deviceBetweenClassVariances);

    // Check for device error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Device error: %s\n", hipGetErrorString(err));
        hipFree(deviceHistogram);
        return 0;
    }

    // Copy interclass variances back to host
	hipMemcpy(hostBetweenClassVariances, deviceBetweenClassVariances, sizeof(double) * MAXPIXVAL, hipMemcpyDeviceToHost);

	hipFree(deviceHistogram);
	hipFree(deviceBetweenClassVariances);

    // Find the highest variance (TODO: Invert this?)
	double maxVariance = 0;
	unsigned char currentBestThreshold = 0;
	for (int t = 0; t < MAXPIXVAL; t++) {
		if (hostBetweenClassVariances[t] > maxVariance) {
			currentBestThreshold = (unsigned char)t;
			maxVariance = hostBetweenClassVariances[t];
		}
	}

	delete hostBetweenClassVariances;

	return currentBestThreshold;
}
